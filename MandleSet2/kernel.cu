#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <iostream>
#include <time.h>
#include <cmath>

#define Height 800
#define Width 800
#define batch 400
#define trials 400
#define rate 0.05
#define ReCoord -0.761574
#define ImCoord -0.0847596
#define Intensity 0.5
#define Speed 1
#define debug false

using namespace cv;
using namespace std;

typedef struct {
	double* N;
	double* I;
	double* posN;
	double* posI;
}Com;

double mapp(double i, double Mid, double Range, int size) {
	return (((i / double(size - 1)) - 0.5) * Range) + Mid;
}

double func(double i, int size, double offset, double magnification) {
	return (((i / double(size - 1)) + offset) / magnification);
}

__global__ void calculate(Com C, uint8_t* Conf) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	C.I[index] = 0;
	C.N[index] = 0;
	double tempN = C.N[index];
	double sqrMagN = tempN * tempN;
	double sqrMagI = C.I[index] * C.I[index];

	Conf[index] = 255;

	int i = 0;
	for (i = 0; i < trials; i++) {
		tempN = C.N[index];

		C.N[index] = (sqrMagN - sqrMagI) + C.posN[index];
		C.I[index] = (2 * C.I[index] * tempN) + C.posI[index];

		sqrMagN = (C.N[index] * C.N[index]);
		sqrMagI = (C.I[index] * C.I[index]);

		if ((sqrMagN + sqrMagI) >= 4) {
			if (i * Intensity < 256)
				Conf[index] = (uint8_t)(i * Intensity);
			else
				Conf[index] = 255;
			break;
		}
	}
}

__global__ void initialize(Com C, int bat, double iteration) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	C.I[index] = 0;
	C.N[index] = 0;
	C.posN[index] = ((((double)threadIdx.x / double(Width - 1)) - 0.5) / iteration) + ReCoord;
	C.posI[index] = ((((double)(bat * batch + blockIdx.x) / double(Height - 1)) - 0.5) / iteration) + ImCoord;

}

void checkDevice() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
}

int main()
{
	checkDevice();

	//Image Variable
	Mat image = Mat::zeros(Height, Width, CV_8UC1);

	//Com C;
	Com c_C;
	uint8_t* Conf;
	uint8_t* c_Conf;

	size_t size = sizeof(double) * Height * batch;

	//C.I = (double*)malloc(size);
	//C.N = (double*)malloc(size);
	Conf = (uint8_t*)malloc(sizeof(uint8_t) * Height * batch);
	hipMalloc(&c_C.I, size);
	hipMalloc(&c_C.N, size);
	hipMalloc(&c_C.posI, size);
	hipMalloc(&c_C.posN, size);
	hipMalloc(&c_Conf, sizeof(uint8_t) * Height * batch);

	int z = 0;
	int iter = 0;

	while (!debug) {
		for (z = 0; z < (Width / batch); z+=Speed) {
			int i = 0;
			int j = 0;

			initialize << <batch, Height >> > (c_C, z, (double)pow(2, (rate * (double)iter)));

			hipDeviceSynchronize();
			
			hipMemcpy(c_Conf, Conf, sizeof(uint8_t) * Height * batch, hipMemcpyHostToDevice);

			calculate << <batch, Height >> > (c_C, c_Conf);

			hipDeviceSynchronize();

			//hipMemcpy(C.I, c_C.I, size, hipMemcpyDeviceToHost);
			//hipMemcpy(C.N, c_C.N, size, hipMemcpyDeviceToHost);
			hipMemcpy(Conf, c_Conf, sizeof(uint8_t) * Height * batch, hipMemcpyDeviceToHost);

			for (j = 0; j < batch; j++) {
				for (i = 0; i < Height; i++) {
					image.data[((z * batch + j) * Height + i)] = Conf[(j * Height + i)];
				}
			}
		}
		
		
		imshow("Display Window", image);
		char c = (char)waitKey(25);
		if (c == 27)
			break;
		

		iter++;
	}

	hipFree(c_C.I);
	hipFree(c_C.N);
	hipFree(c_C.posI);
	hipFree(c_C.posN);
	hipFree(c_Conf);
	free(Conf);
	//free(C.I);
	//free(C.N);
	//free(C.posI);
	//free(C.posN);


	/*
	imshow("Display Window", image);
	waitKey(0);
	*/

	return 0;
}